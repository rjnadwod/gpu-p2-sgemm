/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mysgemm(int m, int n, int k, const float *A, const float *B, float* C) {

    /********************************************************************
     *
     * Compute C = A x B
     *   where A is a (m x k) matrix
     *   where B is a (k x n) matrix
     *   where C is a (m x n) matrix
     *
     ********************************************************************/

    // INSERT KERNEL CODE HERE

    //Compute matrix C
    int ROW = blockIdx.y * blockDim.y + threadIdx.y;
    int COL = blockIdx.x * blockDim.x + threadIdx.x;
    float SUM = 0;

    if (COL < n && ROW < m)
    {
        for(unsigned int i = 0; i < k; ++i)
        {
            SUM += A[ROW * k + i] * B[i * n + COL];
        }
        C[ROW * n + COL] = SUM;
    }
}

void basicSgemm(char transa, char transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    if ((transa != 'N') && (transa != 'n')) {
	printf("unsupported value of 'transa'\n");
    	return;
    }

    if ((transb != 'N') && (transb != 'n')) {
	printf("unsupported value of 'transb'\n");
	return;
    }

    if ((alpha - 1.0f > 1e-10) || (alpha - 1.0f < -1e-10)) {
	printf("unsupported value of alpha\n");
	return;
    }

    if ((beta - 0.0f > 1e-10) || (beta - 0.0f < -1e-10)) {
	printf("unsupported value of beta\n");
	return;
    }

    // Initialize thread block and kernel grid dimensions ---------------------

    const unsigned int BLOCK_SIZE = 16; // Use 16x16 thread blocks

    //INSERT CODE HERE
    unsigned int gridrows = ceil((m + BLOCK_SIZE - 1) / BLOCK_SIZE);
    unsigned int gridcols = ceil((n + BLOCK_SIZE - 1) / BLOCK_SIZE);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(gridcols, gridrows);

    // Invoke CUDA kernel -----------------------------------------------------

    //INSERT CODE HERE
    mysgemm<<<dimGrid, dimBlock>>>(m, n, k, A, B, C);


}
